#include "hip/hip_runtime.h"
#include <list>
#include <math.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <mechsys/flbm/Domain.h>
#include <mechsys/dem/domain.h>
#include <mechsys/util/util.h>
struct UserData
{

    thrust::device_vector<real> Xmin;
    thrust::device_vector<real> Xmax;
    thrust::device_vector<real> Ymin;
    thrust::device_vector<real> Ymax;
    thrust::device_vector<real> Zmin;
    thrust::device_vector<real> Zmax;

    real *pXmin;
    real *pXmax;
    real *pYmin;
    real *pYmax;
    real *pZmin;
    real *pZmax;

    real ome;
    size_t block;
    real Head;
    real Orig;
    real Tf;
    real dtOut;
    real time;
    real rho;
    Vec3_t Dp;
    std::ofstream oss_ss;
};

// Enumeration to define different boundary condition types.
enum BoundaryConditionType
{
    BCT_XMIN0,
    BCT_XMIN1,
    BCT_XMAX0,
    BCT_XMAX1,
    BCT_YMIN0,
    BCT_YMIN1,
    BCT_YMAX0,
    BCT_YMAX1,
    BCT_ZMIN0,
    BCT_ZMIN1,
    BCT_ZMAX0,
    BCT_ZMAX1
};

__global__ void SetupBoundaryConditions(real *rhoBC, bool *IsSolid, real *F, real3 *Vel, real *Rho, FLBM::lbm_aux *lbmaux, BoundaryConditionType bcType)
{

    int ic = threadIdx.x + blockIdx.x * blockDim.x;
    int Nx = lbmaux[0].Nx;
    int Ny = lbmaux[0].Ny;
    int Nz = lbmaux[0].Nz;

    size_t ib, ix, iy, iz, iv;

    switch (bcType)
    {
    case BCT_XMIN0:

        if (ic >= Ny * Nz)
            return;
        ix = 0;
        iy = ic % Ny;
        iz = (ic / Ny) % Nz;
        ib = ix + iy * Nx + iz * Nx * Ny;
        //if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = &F[iv];
            f[1] = 1.0 / 3.0 * (-2 * f[0] - 4 * f[10] - 4 * f[12] - 4 * f[14] - f[2] - 2 * f[3] - 2 * f[4] - 2 * f[5] - 2 * f[6] - 4 * f[8] + 2 * rhoBC[0]);
            f[7] = 1.0 / 24.0 * (-2 * f[0] - 4 * f[10] - 4 * f[12] - 4 * f[14] - 4 * f[2] + f[3] - 5 * f[4] + f[5] - 5 * f[6] + 20 * f[8] + 2 * rhoBC[0]);
            f[9] = 1.0 / 24.0 * (-2 * f[0] + 20 * f[10] - 4 * f[12] - 4 * f[14] - 4 * f[2] + f[3] - 5 * f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[0]);
            f[11] = 1.0 / 24.0 * (-2 * f[0] - 4 * f[10] + 20 * f[12] - 4 * f[14] - 4 * f[2] - 5 * f[3] + f[4] + f[5] - 5 * f[6] - 4 * f[8] + 2 * rhoBC[0]);
            f[13] = 1.0 / 24.0 * (-2 * f[0] - 4 * f[10] - 4 * f[12] + 20 * f[14] - 4 * f[2] - 5 * f[3] + f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[0]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {
                //if ((iz==Nz/2)&&(iy==Ny/2)) printf("F %g %g %d \n",F[iv + k],f[k],k);
                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            //  printf("xmin0  rhoBC[0] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //    rhoBC[0], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);


    // printf("xmin0  rhoBC[0] = %f, Rho[ib+Ny*Nz] = %f\n", rhoBC[0], Rho[ib+Ny*Nz]);

            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];

        }

        break;
    case BCT_XMIN1:
        if (ic >= Ny * Nz)
            return;
        ix = 0;
        iy = ic % Ny;
        iz = (ic / Ny) % Nz;
        ib = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;
        // ib = ix * Nz * Ny + iy * Nz + iz + Nx * Ny * Nz;

        //if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[1] = 1.0 / 3.0 * (-2 * f[0] - 4 * f[10] - 4 * f[12] - 4 * f[14] - f[2] - 2 * f[3] - 2 * f[4] - 2 * f[5] - 2 * f[6] - 4 * f[8] + 2 * rhoBC[1]);
            f[7] = 1.0 / 24.0 * (-2 * f[0] - 4 * f[10] - 4 * f[12] - 4 * f[14] - 4 * f[2] + f[3] - 5 * f[4] + f[5] - 5 * f[6] + 20 * f[8] + 2 * rhoBC[1]);
            f[9] = 1.0 / 24.0 * (-2 * f[0] + 20 * f[10] - 4 * f[12] - 4 * f[14] - 4 * f[2] + f[3] - 5 * f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[1]);
            f[11] = 1.0 / 24.0 * (-2 * f[0] - 4 * f[10] + 20 * f[12] - 4 * f[14] - 4 * f[2] - 5 * f[3] + f[4] + f[5] - 5 * f[6] - 4 * f[8] + 2 * rhoBC[1]);
            f[13] = 1.0 / 24.0 * (-2 * f[0] - 4 * f[10] - 4 * f[12] + 20 * f[14] - 4 * f[2] - 5 * f[3] + f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[1]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("xmin0  rhoBC[1] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[1], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;

    case BCT_XMAX0:
        if (ic >= Ny * Nz)
            return;
        ix = Nx - 1;
        iy = ic % Ny;
        iz = ic / Ny;
        // idx = ix + iy * Nx + iz * Nx * Ny;

        ib = ix + iy * Nx + iz * Nx * Ny;

        //if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[2] = 1 / 3.0 * (-2 * f[0] - f[1] - 2 * (2 * f[11] + 2 * f[13] + f[3] + f[4] + f[5] + f[6] + 2 * f[7] + 2 * f[9] - rhoBC[0]));
            f[8] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] - 4 * f[11] - 4 * f[13] - 5 * f[3] + f[4] - 5 * f[5] + f[6] + 20 * f[7] - 4 * f[9] + 2 * rhoBC[0]);
            f[10] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] - 4 * f[11] - 4 * f[13] - 5 * f[3] + f[4] + f[5] - 5 * f[6] - 4 * f[7] + 20 * f[9] + 2 * rhoBC[0]);
            f[12] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] + 20 * f[11] - 4 * f[13] + f[3] - 5 * f[4] - 5 * f[5] + f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[0]);
            f[14] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] - 4 * f[11] + 20 * f[13] + f[3] - 5 * f[4] + f[5] - 5 * f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[0]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("rhoBC[0] = %f, Rho[ib] = %f, f[0] = %f, f[2] = %f, f[8] = %f, f[10] = %f, f[12] = %f, f[14] = %f\n", rhoBC[0], Rho[ib], f[0], f[2], f[8], f[10], f[12], f[14]);

            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];

        }

        break;
    case BCT_XMAX1:
        if (ic >= Ny * Nz)
            return;
        ix = Nx - 1;
        iy = ic % Ny;
        iz = ic / Ny;
        // idx = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;

        ib = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;

        //if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[2] = 1 / 3.0 * (-2 * f[0] - f[1] - 2 * (2 * f[11] + 2 * f[13] + f[3] + f[4] + f[5] + f[6] + 2 * f[7] + 2 * f[9] - rhoBC[1]));
            f[8] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] - 4 * f[11] - 4 * f[13] - 5 * f[3] + f[4] - 5 * f[5] + f[6] + 20 * f[7] - 4 * f[9] + 2 * rhoBC[1]);
            f[10] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] - 4 * f[11] - 4 * f[13] - 5 * f[3] + f[4] + f[5] - 5 * f[6] - 4 * f[7] + 20 * f[9] + 2 * rhoBC[1]);
            f[12] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] + 20 * f[11] - 4 * f[13] + f[3] - 5 * f[4] - 5 * f[5] + f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[1]);
            f[14] = 1 / 24.0 * (-2 * f[0] - 4 * f[1] - 4 * f[11] + 20 * f[13] + f[3] - 5 * f[4] + f[5] - 5 * f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[1]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }

            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;
    case BCT_YMIN0:
        if (ic >= Nx * Nz)
            return;
        ix = ic % Nx;
        iy = 0;
        iz = ic / Nx;
        // idx = ix + iy * Nx + iz * Nx * Ny;
        ib = ix + iy * Nx + iz * Nx * Ny;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;

            f[3] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[10] - 4 * f[11] - 4 * f[13] - 2 * f[2] - f[4] - 2 * f[5] - 2 * f[6] - 4 * f[8] + 2 * rhoBC[0]);
            f[7] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[10] - 4 * f[11] - 4 * f[13] - 5 * f[2] - 4 * f[4] + f[5] - 5 * f[6] + 20 * f[8] + 2 * rhoBC[0]);
            f[9] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[10] - 4 * f[11] - 4 * f[13] - 5 * f[2] - 4 * f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[0]);
            f[12] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] + 20 * f[11] - 4 * f[13] + f[2] - 4 * f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[0]);
            f[14] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] - 4 * f[11] + 20 * f[13] + f[2] - 4 * f[4] + f[5] - 5 * f[6] - 4 * f[8] + 2 * rhoBC[0]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("YMIN0  rhoBC[0] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[0], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;
    case BCT_YMIN1:
        if (ic >= Nx * Nz)
            return;
        ix = ic % Nx;
        iy = 0;
        iz = ic / Nx;
        ib = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[3] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[10] - 4 * f[11] - 4 * f[13] - 2 * f[2] - f[4] - 2 * f[5] - 2 * f[6] - 4 * f[8] + 2 * rhoBC[1]);
            f[7] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[10] - 4 * f[11] - 4 * f[13] - 5 * f[2] - 4 * f[4] + f[5] - 5 * f[6] + 20 * f[8] + 2 * rhoBC[1]);
            f[9] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[10] - 4 * f[11] - 4 * f[13] - 5 * f[2] - 4 * f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[1]);
            f[12] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] + 20 * f[11] - 4 * f[13] + f[2] - 4 * f[4] - 5 * f[5] + f[6] - 4 * f[8] + 2 * rhoBC[1]);
            f[14] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] - 4 * f[11] + 20 * f[13] + f[2] - 4 * f[4] + f[5] - 5 * f[6] - 4 * f[8] + 2 * rhoBC[1]);
            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("YMIN1  rhoBC[1] = %f, Rho[ib] = %f, F + iv,f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[1], Rho[ib], F[iv ],f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;
    case BCT_YMAX0:
        if (ic >= Nx * Nz)
            return;
        ix = ic % Nx;
        iy = Ny - 1;
        iz = ic / Nx;

        ib = ix + iy * Nx + iz * Nx * Ny;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[4] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[12] - 4 * f[14] - 2 * f[2] - f[3] - 2 * f[5] - 2 * f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[0]);
            f[8] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] - 4 * f[14] + f[2] - 4 * f[3] - 5 * f[5] + f[6] + 20 * f[7] - 4 * f[9] + 2 * rhoBC[0]);
            f[10] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] - 4 * f[14] + f[2] - 4 * f[3] + f[5] - 5 * f[6] - 4 * f[7] + 20 * f[9] + 2 * rhoBC[0]);
            f[11] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[12] - 4 * f[14] - 5 * f[2] - 4 * f[3] + f[5] - 5 * f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[0]);
            f[13] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[12] + 20 * f[14] - 5 * f[2] - 4 * f[3] - 5 * f[5] + f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[0]);
            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("YMMAX0  rhoBC[0] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[0], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;
    case BCT_YMAX1:
        if (ic >= Nx * Nz)
            return;
        ix = ic % Nx;
        iy = Ny - 1;
        iz = ic / Nx;
        // idx = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;
        ib = ix + iy * Nx + iz * Nx * Ny + lbmaux[0].Nx * lbmaux[0].Ny * lbmaux[0].Nz;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[4] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[12] - 4 * f[14] - 2 * f[2] - f[3] - 2 * f[5] - 2 * f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[1]);
            f[8] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] - 4 * f[14] + f[2] - 4 * f[3] - 5 * f[5] + f[6] + 20 * f[7] - 4 * f[9] + 2 * rhoBC[1]);
            f[10] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] - 4 * f[14] + f[2] - 4 * f[3] + f[5] - 5 * f[6] - 4 * f[7] + 20 * f[9] + 2 * rhoBC[1]);
            f[11] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[12] - 4 * f[14] - 5 * f[2] - 4 * f[3] + f[5] - 5 * f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[1]);
            f[13] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[12] + 20 * f[14] - 5 * f[2] - 4 * f[3] - 5 * f[5] + f[6] - 4 * f[7] - 4 * f[9] + 2 * rhoBC[1]);
            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("Ymax1  rhoBC[1] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[1], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;
    case BCT_ZMIN0:
        if (ic >= Nx * Ny)
            return;
        ix = ic % Nx;
        iy = (ic / Nx) % Ny;
        iz = 0;
        //  idx = ix + iy * Nx + iz * Nx * Ny;
        ib = ix + iy * Nx + iz * Nx * Ny;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[5] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[12] - 4 * f[13] - 2 * f[2] - 2 * f[3] - 2 * f[4] - f[6] - 4 * f[8] - 4 * f[9] + 2 * rhoBC[0]);
            f[7] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[12] - 4 * f[13] - 5 * f[2] + f[3] - 5 * f[4] - 4 * f[6] + 20 * f[8] - 4 * f[9] + 2 * rhoBC[0]);
            f[10] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] - 4 * f[13] + f[2] - 5 * f[3] + f[4] - 4 * f[6] - 4 * f[8] + 20 * f[9] + 2 * rhoBC[0]);
            f[11] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[12] - 4 * f[13] - 5 * f[2] - 5 * f[3] + f[4] - 4 * f[6] - 4 * f[8] - 4 * f[9] + 2 * rhoBC[0]);
            f[14] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] + 20 * f[13] + f[2] + f[3] - 5 * f[4] - 4 * f[6] - 4 * f[8] - 4 * f[9] + 2 * rhoBC[0]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("ZMIN)  rhoBC[0] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[0], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }

        break;
    case BCT_ZMIN1:
        if (ic >= Nx * Ny)
            return;
        ix = ic % Nx;
        iy = (ic / Nx) % Ny;
        iz = 0;

        ib = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[5] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[12] - 4 * f[13] - 2 * f[2] - 2 * f[3] - 2 * f[4] - f[6] - 4 * f[8] - 4 * f[9] + 2 * rhoBC[1]);
            f[7] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[12] - 4 * f[13] - 5 * f[2] + f[3] - 5 * f[4] - 4 * f[6] + 20 * f[8] - 4 * f[9] + 2 * rhoBC[1]);
            f[10] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] - 4 * f[13] + f[2] - 5 * f[3] + f[4] - 4 * f[6] - 4 * f[8] + 20 * f[9] + 2 * rhoBC[1]);
            f[11] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[12] - 4 * f[13] - 5 * f[2] - 5 * f[3] + f[4] - 4 * f[6] - 4 * f[8] - 4 * f[9] + 2 * rhoBC[1]);
            f[14] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[12] + 20 * f[13] + f[2] + f[3] - 5 * f[4] - 4 * f[6] - 4 * f[8] - 4 * f[9] + 2 * rhoBC[1]);

            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("ZMIN1  rhoBC[1 = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[0], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];

            // printf("zmin1   f[0]= %f , f[1] =%f, f[2] =%f, f[3] =%f, f[4] =%f, f[5] =%f, f[6] =%f, f[7] =%f, f[8] =%f, f[9] =%f, f[10] =%f, f[11] =%f, f[12] =%f, f[13] =%f, f[14] =%f\n", f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
        }
        break;
    case BCT_ZMAX0:
        if (ic >= Nx * Ny)
            return;
        ix = ic % Nx;
        iy = (ic / Nx) % Ny;
        iz = Nz - 1;

        ib = ix + iy * Nx + iz * Nx * Ny;

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[6] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[10] - 4 * f[11] - 4 * f[14] - 2 * f[2] - 2 * f[3] - 2 * f[4] - f[5] - 4 * f[7] + 2 * rhoBC[0]);
            f[8] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] - 4 * f[11] - 4 * f[14] + f[2] - 5 * f[3] + f[4] - 4 * f[5] + 20 * f[7] + 2 * rhoBC[0]);
            f[9] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[10] - 4 * f[11] - 4 * f[14] - 5 * f[2] + f[3] - 5 * f[4] - 4 * f[5] - 4 * f[7] + 2 * rhoBC[0]);
            f[12] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] + 20 * f[11] - 4 * f[14] + f[2] + f[3] - 5 * f[4] - 4 * f[5] - 4 * f[7] + 2 * rhoBC[0]);
            f[13] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[10] - 4 * f[11] + 20 * f[14] - 5 * f[2] - 5 * f[3] + f[4] - 4 * f[5] - 4 * f[7] + 2 * rhoBC[0]);
            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            //  printf("ZMAX0  rhoBC[0] = %f, Rho[ib] = %f, F + iv,f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //                    rhoBC[0], Rho[ib], F[iv ],f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);

            // printf("ZMax0  rhoBC[0] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[0], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;

    case BCT_ZMAX1:
        if (ic >= Nx * Ny)
            return;
        ix = ic % Nx;
        iy = (ic / Nx) % Ny;
        iz = Nz - 1;
        ib = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz;
        // idx = ix + iy * Nx + iz * Nx * Ny + Nx * Ny * Nz; // this is the idx for fluid 1 and there is no lbmaux[1]

        if (!IsSolid[ib])
        {
            size_t iv = ib * lbmaux[0].Nneigh;
            real *f = F + iv;
            f[6] = 1 / 3.0 * (-2 * f[0] - 2 * f[1] - 4 * f[10] - 4 * f[11] - 4 * f[14] - 2 * f[2] - 2 * f[3] - 2 * f[4] - f[5] - 4 * f[7] + 2 * rhoBC[1]);
            f[8] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] - 4 * f[11] - 4 * f[14] + f[2] - 5 * f[3] + f[4] - 4 * f[5] + 20 * f[7] + 2 * rhoBC[1]);
            f[9] = 1 / 24.0 * (-2 * f[0] + f[1] + 20 * f[10] - 4 * f[11] - 4 * f[14] - 5 * f[2] + f[3] - 5 * f[4] - 4 * f[5] - 4 * f[7] + 2 * rhoBC[1]);
            f[12] = 1 / 24.0 * (-2 * f[0] - 5 * f[1] - 4 * f[10] + 20 * f[11] - 4 * f[14] + f[2] + f[3] - 5 * f[4] - 4 * f[5] - 4 * f[7] + 2 * rhoBC[1]);
            f[13] = 1 / 24.0 * (-2 * f[0] + f[1] - 4 * f[10] - 4 * f[11] + 20 * f[14] - 5 * f[2] - 5 * f[3] + f[4] - 4 * f[5] - 4 * f[7] + 2 * rhoBC[1]);
            Rho[ib] = 0.0;
            Vel[ib] = make_real3(0.0, 0.0, 0.0);
            for (size_t k = 0; k < lbmaux[0].Nneigh; k++)
            {

                Rho[ib] += F[iv + k];
                Vel[ib] = Vel[ib] + F[iv + k] * lbmaux[0].C[k];
            }
            // printf("ZMAX1  rhoBC[1] = %f, Rho[ib] = %f, f[0] = %f, f[1] = %f, f[2] = %f, f[3] = %f, f[4] = %f, f[5] = %f, f[6] = %f, f[7] = %f, f[8] = %f, f[9] = %f, f[10] = %f, f[11] = %f, f[12] = %f, f[13] = %f, f[14] = %f\n",
            //        rhoBC[1], Rho[ib], f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14]);
            Vel[ib] = (lbmaux[0].Cs / Rho[ib]) * Vel[ib];
        }
        break;
    }
}
void Setup(FLBM::Domain &dom, void *UD)
{

    UserData &dat = (*static_cast<UserData *>(UD));
    if (dom.Time > dat.time)
    {
        dat.time += dat.dtOut;
    }
    real a = M_PI / dat.ome;
    real rho = dat.Head * ((1.0 / a) * (dat.time - a * (floor(dat.time / a) + 0.5)) * pow(-1.0, floor(dat.time / a)) + 0.5) + dat.Orig;
    real rho0min;
    real rho1min;
    real rho0max;
    real rho1max;
    int sizeX = dom.Ndim(0);
    int sizeY = dom.Ndim(1);
    int sizeZ = dom.Ndim(2);

    real *pXmin = thrust::raw_pointer_cast(dat.Xmin.data());
    real *pXmax = thrust::raw_pointer_cast(dat.Xmax.data());
    real *pYmin = thrust::raw_pointer_cast(dat.Ymin.data());
    real *pYmax = thrust::raw_pointer_cast(dat.Ymax.data());
    real *pZmin = thrust::raw_pointer_cast(dat.Zmin.data());
    real *pZmax = thrust::raw_pointer_cast(dat.Zmax.data());

    if (fabs(dat.Dp(0)) > 1.0e-12)
    {
        rho0min = 0.999 * ((rho - dat.rho) * dat.Dp(0) + dat.rho);
        rho1min = 0.001 * ((rho - dat.rho) * dat.Dp(0) + dat.rho);
        rho0max = 0.001 * ((dat.rho - rho) * dat.Dp(0) + dat.rho);
        rho1max = 0.999 * ((dat.rho - rho) * dat.Dp(0) + dat.rho);

        // dat.Xmin[0] = 20.0;

        dat.Xmin[0] = rho0min;

        SetupBoundaryConditions<<<(dom.Ndim(1) * dom.Ndim(2)) / dom.Nthread + 1, dom.Nthread>>>(pXmin, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_XMIN0);
        hipDeviceSynchronize();

        dat.Xmax[0] = rho0max;
        SetupBoundaryConditions<<<(dom.Ndim(1) * dom.Ndim(2)) / dom.Nthread + 1, dom.Nthread>>>(pXmax, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_XMAX0);
        hipDeviceSynchronize();

        dat.Xmin[1] = rho1min;
        SetupBoundaryConditions<<<(dom.Ndim(1) * dom.Ndim(2)) / dom.Nthread + 1, dom.Nthread>>>(pXmin, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_XMIN1);
        hipDeviceSynchronize();

        dat.Xmax[1] = rho1max;
        SetupBoundaryConditions<<<(dom.Ndim(1) * dom.Ndim(2)) / dom.Nthread + 1, dom.Nthread>>>(pXmax, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_XMAX1);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        }
    }

    if (fabs(dat.Dp(1)) > 1.0e-12)
    {
        dat.Ymin[0] = rho0min = 0.999 * ((rho - dat.rho) * dat.Dp(1) + dat.rho);
        dat.Ymin[1] = rho1min = 0.001 * ((rho - dat.rho) * dat.Dp(1) + dat.rho);
        dat.Ymax[0] = rho0max = 0.001 * ((dat.rho - rho) * dat.Dp(1) + dat.rho);
        dat.Ymax[1] = rho1max = 0.999 * ((dat.rho - rho) * dat.Dp(1) + dat.rho);

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(2) / dom.Nthread + 1, dom.Nthread>>>(pYmin, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_YMIN0);
        hipDeviceSynchronize();

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(2) / dom.Nthread + 1, dom.Nthread>>>(pYmax, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_YMAX0);
        hipDeviceSynchronize();

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(2) / dom.Nthread + 1, dom.Nthread>>>(pYmin, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_YMIN1);
        hipDeviceSynchronize();

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(2) / dom.Nthread + 1, dom.Nthread>>>(pYmax, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_YMAX1);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        }
    }

    if (fabs(dat.Dp(2)) > 1.0e-12)
    {
        dat.Zmin[0] = rho0min = 0.999 * ((rho - dat.rho) * dat.Dp(2) + dat.rho);
        dat.Zmin[1] = rho1min = 0.001 * ((rho - dat.rho) * dat.Dp(2) + dat.rho);
        dat.Zmax[0] = rho0max = 0.001 * ((dat.rho - rho) * dat.Dp(2) + dat.rho);
        dat.Zmax[1] = rho1max = 0.999 * ((dat.rho - rho) * dat.Dp(2) + dat.rho);

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(1) / dom.Nthread + 1, dom.Nthread>>>(pZmin, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_ZMIN0);
        hipDeviceSynchronize();

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(1) / dom.Nthread + 1, dom.Nthread>>>(pZmax, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_ZMAX0);
        hipDeviceSynchronize();

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(1) / dom.Nthread + 1, dom.Nthread>>>(pZmin, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_ZMIN1);
        hipDeviceSynchronize();

        SetupBoundaryConditions<<<dom.Ndim(0) * dom.Ndim(1) / dom.Nthread + 1, dom.Nthread>>>(pZmax, dom.pIsSolid, dom.pF, dom.pVel, dom.pRho, dom.plbmaux, BCT_ZMAX1);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        }
    }
}

void Report(FLBM::Domain &dom, void *UD)
{
    UserData &dat = (*static_cast<UserData *>(UD));
    double water = 0.0;
    double oil = 0.0;
    double Sr = 0.0;
    size_t nw = 0;
    size_t no = 0;

    for (size_t idx = 0; idx < dom.Ncells; idx++)
    {
        iVec3_t coord;
        FLBM::idx2Pt(idx, coord, dom.Ndim);
        bool isSolid = dom.IsSolid[0][coord(0)][coord(1)][coord(2)];
        if (!isSolid)
        {
            double wr = dom.Rho[1][coord(0)][coord(1)][coord(2)];
            double ar = dom.Rho[0][coord(0)][coord(1)][coord(2)];
            if (wr > 0.5 * dat.rho)
            {
                Sr += 1.0;
                water += (wr + ar + dom.Gmix * wr * ar) / 3.0;
                nw++;
            }
            if (ar > 0.5 * dat.rho)
            {
                oil += (wr + ar + dom.Gmix * wr * ar) / 3.0;
                no++;
            }
        }
    }

    double Sf = 0.0;

    for (size_t x = 0; x < dom.Ndim(0); x++)
    {
        for (size_t y = 0; y < dom.Ndim(1); y++)
        {
            for (size_t z = 0; z < dom.Ndim(2); z++)
            {
                if (dom.IsSolid[0][x][y][z])
                {
                    Sf += 1.0;
                }
            }
        }
    } // Solid fraction，即固体分数

    double aaa = Sf / dom.Ncells;

    Sr /= dom.Ncells * (1.0 - aaa);
    if (nw > 0)
        water /= nw;
    if (no > 0)
        oil /= no; //  果然是这里计算有问题

    //   oil = no;
    double rhow = 0.0;
    double rhoo = 0.0;
    size_t nfb = 0;
    size_t nfo = 0;

    for (size_t i = 0; i < dom.Ndim(1); ++i)
    {
        for (size_t j = 0; j < dom.Ndim(2); ++j)
        {
            // 检查边界的固体状态
            if (!dom.IsSolid[0][1][i][j])
            {
                double rho = dom.Rho[0][1][i][j]; //  0 phase  0.999*2
                rhow += rho;
                nfb++;
          //   printf("dom.Rho[0][1][%lu][%lu]=%f\n",i,j, dom.Rho[0][1][i][j]);

            }

            if (!dom.IsSolid[1][dom.Ndim(0) - 2][i][j])
            {
                double rho = dom.Rho[1][dom.Ndim(0) - 2][i][j]; // 同上
                rhoo += rho;
                nfo++;
            }
        }
    }

  
    
    // if (nfb > 0)
    rhow /= nfb;
    //   if (nfo > 0)
    rhoo /= nfo;

    double Pc;
    double rho;
     size_t newSum = (nw+no);
    double newSr = (1.00*nw)/newSum;
    double a = M_PI / dat.ome;
    rho = dat.Head * ((1.0 / a) * (dat.time - a * (floor(dat.time / a) + 0.5)) * pow(-1.0, floor(dat.time / a)) + 0.5) + dat.Orig;
    Pc = (2.0 * (rho - dat.rho) + dom.Gmix * (rho * rho * 0.999 * 0.001 - (2.0 * dat.rho - rho) * (2.0 * dat.rho - rho) * 0.999 * 0.001)) / 3.0;
   // dat.oss_ss << dom.Time  << Util::_8s << "nw"<< Util::_8s << nw << Util::_8s << "no"<< Util::_8s << no << Util::_8s << "Pc" << Pc << Util::_8s << Util::_8s << "Sr = nw/(nw+no)"<< newSr<< Util::_8s << "newSum = (nw+no)"<< Util::_8s << newSum << Util::_8s << "Sr-newSr"<< Util::_8s << Sr-newSr << Util::_8s << "Ncells- Sf"<< dom.Ncells-Sf<< std::endl;
    
    dat.oss_ss << dom.Time << Util::_8s << rho << Util::_8s << rhoo << Util::_8s << rhow << Util::_8s << water << Util::_8s << oil << Util::_8s << Pc << Util::_8s << Sr << std::endl;
}

int main(int argc, char **argv)
try
{
    String filekey(argv[1]);
    String filename(filekey + ".inp");
    if (!Util::FileExists(filename))
        throw new Fatal("File <%s> not found", filename.CStr());
    std::ifstream infile(filename.CStr());
    size_t Nproc = 1;
    if (argc == 3)
        Nproc = atoi(argv[2]);

    String fileDEM;
    String fileLBM;
    bool Render = true;
    size_t N = 200;
    real Gs0 = -0.53;
    real Gs1 = -0.53;
    real Gmix = 2.0;
    double nu = 0.05;
    double dt = 1.0;
    double Tf = 10000.0;
    real dtOut = 50.0;
    real HeadStep = 1000.0;
    real rho = 200.0;
    real ome = 2.0;
    real Head = 500.0;
    real Orig = 54.0;
    size_t oct = 1;
    real DPx = 1.0;
    real DPy = 1.0;
    real DPz = 1.0;
    int outlimit = 1;
    size_t buffer = 1;
    {
        infile >> fileDEM;
        infile.ignore(200, '\n');
        infile >> fileLBM;
        infile.ignore(200, '\n');
        infile >> Render;
        infile.ignore(200, '\n');
        infile >> N;
        infile.ignore(200, '\n');
        infile >> Gs0;
        infile.ignore(200, '\n');
        infile >> Gs1;
        infile.ignore(200, '\n');
        infile >> Gmix;
        infile.ignore(200, '\n');
        infile >> nu;
        infile.ignore(200, '\n');
        infile >> dt;
        infile.ignore(200, '\n');
        infile >> Tf;
        infile.ignore(200, '\n');
        infile >> dtOut;
        infile.ignore(200, '\n');
        infile >> HeadStep;
        infile.ignore(200, '\n');
        infile >> rho;
        infile.ignore(200, '\n');
        infile >> ome;
        infile.ignore(200, '\n');
        infile >> Head;
        infile.ignore(200, '\n');
        infile >> Orig;
        infile.ignore(200, '\n');
        infile >> oct;
        infile.ignore(200, '\n');
        infile >> DPx;
        infile.ignore(200, '\n');
        infile >> DPy;
        infile.ignore(200, '\n');
        infile >> DPz;
        infile.ignore(200, '\n');
        infile >> outlimit;
        infile.ignore(200, '\n');
        infile >> buffer;
        infile.ignore(200, '\n');
    }
    Array<real> nua(2);
    nua[0] = nu;
    nua[1] = nu;

    DEM::Domain DemDom;
    DemDom.Load(fileDEM.CStr());
    Array<int> idx(6);
    idx = -2, -3, -4, -5, -6, -7;
    DemDom.DelParticles(idx);
    Vec3_t Xmin, Xmax;
    DemDom.BoundingBox(Xmin, Xmax);
    int bound = outlimit;
    real dx = (Xmax(0) - Xmin(0)) / (N - 2 * bound);
    size_t Ny = (Xmax(1) - Xmin(1)) / dx + 2 * bound;
    size_t Nz = (Xmax(2) - Xmin(2)) / dx + 2 * bound;
    DemDom.Center(0.5 * (Xmax - Xmin) + Vec3_t(bound * dx, bound * dx, bound * dx));

    FLBM::Domain Dom(D3Q15, nua, iVec3_t(N, Ny, Nz), 1.0, 1.0);
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < Ny; y++)
        {
            for (int z = 0; z < Nz; z++)
            {
                Vec3_t pos((x + 0.5) * dx, (y + 0.5) * dx, (z + 0.5) * dx);
                for (DEM::Particle *P : DemDom.Particles)
                {
                    if (P->IsInsideAlt(pos))
                    {
                        Dom.IsSolid[0][x][y][z] = true;
                        Dom.IsSolid[1][x][y][z] = true;
                    }
                }
            }
        }
    }
    UserData dat;
    Dom.UserData = &dat;

    dat.Tf = Tf;
    dat.ome = 2 * M_PI * ome / Tf;
    dat.Orig = Orig;
    dat.dtOut = HeadStep;
    dat.time = 0.0;
    dat.rho = rho;
    dat.Head = Head;
    dat.Dp = Vec3_t(DPx, DPy, DPz);
    dat.Dp /= norm(dat.Dp);
    dat.block = oct;

    Dom.G[0] = 0.0;
    Dom.Gs[0] = Gs0;
    Dom.G[1] = 0.0;
    Dom.Gs[1] = Gs1;
    Dom.Gmix = Gmix;
    dat.Xmin.resize(2);
    dat.Xmax.resize(2);
    dat.Ymin.resize(2);
    dat.Ymax.resize(2);
    dat.Zmin.resize(2);
    dat.Zmax.resize(2);
    // The 6 faces (x,y) (x,z) (y,z) of the cube correspond to 2 of each group


    for (int i = 0; i < N; i++)
    {
        Dom.IsSolid[0][i][0][0] = true;
        Dom.IsSolid[0][i][Ny - 1][0] = true;
        Dom.IsSolid[0][i][0][Nz - 1] = true;
        Dom.IsSolid[0][i][Ny - 1][Nz - 1] = true;
        Dom.IsSolid[1][i][0][0] = true;
        Dom.IsSolid[1][i][Ny - 1][0] = true;
        Dom.IsSolid[1][i][0][Nz - 1] = true;
        Dom.IsSolid[1][i][Ny - 1][Nz - 1] = true;
    }

    for (int i = 0; i < Ny; i++)
    {
        Dom.IsSolid[0][0][i][0] = true;
        Dom.IsSolid[0][N - 1][i][0] = true;
        Dom.IsSolid[0][0][i][Nz - 1] = true;
        Dom.IsSolid[0][N - 1][i][Nz - 1] = true;
        Dom.IsSolid[1][0][i][0] = true;
        Dom.IsSolid[1][N - 1][i][0] = true;
        Dom.IsSolid[1][0][i][Nz - 1] = true;
        Dom.IsSolid[1][N - 1][i][Nz - 1] = true;
    }

    for (int i = 0; i < Nz; i++)
    {
        Dom.IsSolid[0][0][0][i] = true;
        Dom.IsSolid[0][N - 1][0][i] = true;
        Dom.IsSolid[0][0][Ny - 1][i] = true;
        Dom.IsSolid[0][N - 1][Ny - 1][i] = true;
        Dom.IsSolid[1][0][0][i] = true;
        Dom.IsSolid[1][N - 1][0][i] = true;
        Dom.IsSolid[1][0][Ny - 1][i] = true;
        Dom.IsSolid[1][N - 1][Ny - 1][i] = true;
    }

    bound = buffer;
    for (size_t ix = 0; ix < Dom.Ndim(0); ix++)
    {
        for (size_t iy = 0; iy < Dom.Ndim(1); iy++)
        {
            for (size_t iz = 0; iz < Dom.Ndim(2); iz++)
            {
                size_t il0 = 0;
                size_t il1 = 1;
                iVec3_t idx(ix, iy, iz);

                Dom.Initialize(il0, idx, 0.001 * rho, OrthoSys::O);
                Dom.Initialize(il1, idx, 0.999 * rho, OrthoSys::O);
                // if (oct < 2) // inp=>  oct=1

                {
                    //if ((dat.Dp(0) > 1.0e-12) && (ix < Dom.Ndim(0) / bound)) // 100     //change to 2
                    if ((dat.Dp(0) > 1.0e-12) && (ix < bound))
                    {
                        Dom.Initialize(il0, idx, 0.999 * rho, OrthoSys::O);
                        Dom.Initialize(il1, idx, 0.001 * rho, OrthoSys::O);
                    }
                   // if ((dat.Dp(1) > 1.0e-12) && (iy < Dom.Ndim(1) / bound)) // 90
                    if ((dat.Dp(1) > 1.0e-12) && (iy < bound)) // 90
                    {
                        Dom.Initialize(il0, idx, 0.999 * rho, OrthoSys::O);
                        Dom.Initialize(il1, idx, 0.001 * rho, OrthoSys::O);
                    }
                    //if ((dat.Dp(2) > 1.0e-12) && (iz < Dom.Ndim(2) / bound)) // 93
                     if ((dat.Dp(2) > 1.0e-12) && (iz < bound)) // 93
                    {
                        Dom.Initialize(il0, idx, 0.999 * rho, OrthoSys::O);
                        Dom.Initialize(il1, idx, 0.001 * rho, OrthoSys::O);
                    }
                    /*

                                        std::string output;
                                        for (int k = 0; k < 15; k++)
                                        {
                                            output += "F[1][ix][iy][iz][" + std::to_string(k) + "] = " + std::to_string(Dom.F[1][0][0][0][k]) + "\n";
                                        }

                                        printf("%s", output.c_str());
                    */
                }
            }
        }
    }

    // Dom.WriteXDMF("wreten");

    String fs;
    fs.Printf("water_retention.res");
    dat.oss_ss.open(fs.CStr(), std::ios::out);
    dat.oss_ss << Util::_10_6 << "Time" << Util::_8s << "PDen" << Util::_8s << "Rhow" << Util::_8s << "Rhoo" << Util::_8s << "Water" << Util::_8s << "Oil" << Util::_8s << "Pc" << Util::_8s << "Sr" << std::endl;
    Dom.Solve(Tf, dtOut, Setup, Report, filekey.CStr(), Render, Nproc);
    //Dom.Solve(Tf, dtOut, NULL, Report, filekey.CStr(), Render, Nproc);
    dat.oss_ss.close();
}
MECHSYS_CATCH
